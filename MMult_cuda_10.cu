#include <hipblas.h>
#include <hip/hip_runtime.h>

#define SMEM_STRIDE 128

__device__ __forceinline__ uint32_t smem_addr_to_u32(const void *smem_ptr)
{
    uint32_t addr;
    asm("{.reg .u64 u64addr;\n"
        "cvta.to.shared.u64 u64addr, %1;\n"
        "cvt.u32.u64 %0, u64addr;\n}"
        : "=r"(addr)
        : "l"(smem_ptr));
    return addr;
}

__device__ __forceinline__ void ld_global_to_reg_32(float &reg, const void *ptr)
{
    asm volatile("{.reg .pred p;\n"
                 "mov.b32 %0, 0;\n"
#if __CUDA_API_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && __CUDA_ARCH__ >= 750
                 "ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
                 "ld.global.nc.f32 %0, [%1];}\n"
#endif
                 : "=f"(reg)
                 : "l"(ptr));
}

__device__ __forceinline__ void lds128(float &reg0, float &reg1, float &reg2, float &reg3, const uint32_t &addr)
{
    asm volatile("ld.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
                 : "=f"(reg0), "=f"(reg1), "=f"(reg2), "=f"(reg3)
                 : "r"(addr));
}

__device__ __forceinline__ void sts128(const float &reg0, const float &reg1, const float &reg2, const float &reg3, const uint32_t addr)
{
    asm volatile("st.shared.v4.f32 [%0], {%1, %2, %3, %4};\n"
                 :
                 : "r"(addr), "f"(reg0), "f"(reg1), "f"(reg2), "f"(reg3));
}

__device__ __forceinline__ void sts32(const float &reg, const uint32_t addr)
{
    asm volatile("st.shared.f32 [%0], %1;\n"
                 :
                 : "r"(addr), "f"(reg));
}

template <int BLOCK>
__global__ __launch_bounds__(256, 2) void sgemm(int n, int m, int k, float *a, float *b, float *c)
{
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    __shared__ __align__(16 * 1024) char smem[1024 * 24];
    float *ashare = reinterpret_cast<float *>(smem);
    float *bshare = reinterpret_cast<float *>(smem + 16 * 1024);
    float sum[8][8] = {0.f};
    float a_reg[4] = {0.f};
    float b_reg[4] = {0.f};
    float a_plane[8] = {0.f};
    float b_plane[8] = {0.f};

    int a_from = by * 128 * k + tx / 8 * 4 * k + tx % 8;
    int b_from = bx * 128 + tx / 32 * n + tx % 32;

    float *a_from_ptr = a + a_from;
    float *b_from_ptr = b + b_from;
    uint32_t a_to_ptr = smem_addr_to_u32(ashare + tx / 8 * 4 + tx % 8 * SMEM_STRIDE);
    uint32_t b_to_ptr = smem_addr_to_u32(bshare + tx / 32 * SMEM_STRIDE + tx % 32);

    uint32_t a_share_from_ptr = smem_addr_to_u32(ashare + tx / 16 * 4);
    uint32_t b_share_from_ptr = smem_addr_to_u32(bshare + tx % 16 * 4);

    for (int kk = 0; kk < k; kk += 8)
    {
        for (int i = 0; i < 4; ++i)
        {
            ld_global_to_reg_32(a_reg[i], (const char *)(a_from_ptr) + i * k * sizeof(float));
        }
        sts128(a_reg[0], a_reg[1], a_reg[2], a_reg[3], a_to_ptr);

        for (int i = 0; i < 4; ++i)
        {
            ld_global_to_reg_32(b_reg[i], (const char *)(b_from_ptr) + i * 32 * sizeof(float));
        }
        for (int i = 0; i < 4; ++i)
        {
            sts32(b_reg[i], b_to_ptr + i * sizeof(float) * 32);
        }
        __syncthreads();

        a_from_ptr += 8;
        b_from_ptr += 8 * n;

        for (int line = 0; line < 8; ++line)
        {
            lds128(a_plane[0], a_plane[1], a_plane[2], a_plane[3], 
                    a_share_from_ptr + line * SMEM_STRIDE * sizeof(float));
            lds128(a_plane[4], a_plane[5], a_plane[6], a_plane[7], 
                    a_share_from_ptr + (64 + line * SMEM_STRIDE) * sizeof(float));
            lds128(b_plane[0], b_plane[1], b_plane[2], b_plane[3], 
                    b_share_from_ptr + line * SMEM_STRIDE * sizeof(float));
            lds128(b_plane[4], b_plane[5], b_plane[6], b_plane[7],
                    b_share_from_ptr + (64 + line * SMEM_STRIDE) * sizeof(float));

            for (int i = 0; i < 8; ++i)
            {
                for (int j = 0; j < 8; ++j)
                {
                    sum[i][j] += a_plane[i] * b_plane[j];
                }
            }
        }
        __syncthreads();
    }

    float *c_to = c + by * BLOCK * n + bx * BLOCK + (tx / 16 * 4) * n + (tx % 16 * 4);
    for (int i = 0; i < 4; ++i)
    {
        for (int j = 0; j < 4; ++j)
        {
            c_to[i * n + j] = sum[i][j];
            c_to[i * n + j + 64] = sum[i][j + 4];
            c_to[(i + 64) * n + j] = sum[i + 4][j];
            c_to[(i + 64) * n + j + 64] = sum[i + 4][j + 4];
        }
    }
}

void MY_MMult(cublasContext *handle, int n, int m, int k, float *a, int len_a, float *b, int len_b, float *c, int len_c)
{
    constexpr int BLOCK = 128;
    dim3 block(BLOCK * 2);
    dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);
    sgemm<BLOCK><<<grid, block>>>(n, m, k, a, b, c);
}